
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mm_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[64];
  float4 A_local[1];
  float4 B_local[32];
  __shared__ float A_shared[2048];
  __shared__ float B_shared[2048];
  float B_shared_local[8];
  float A_shared_local[8];
  float B_shared_local1[8];
  float A_shared_local1[8];
  for (int i0_2_init = 0; i0_2_init < 8; ++i0_2_init) {
    for (int i1_2_init = 0; i1_2_init < 8; ++i1_2_init) {
      Y_local[((i0_2_init * 8) + i1_2_init)] = 0.000000e+00f;
    }
  }
  A_local[0] = *(float4*)(A + ((((((int)blockIdx.x) * 131072) + (((int)threadIdx.y) * 8192)) + ((((int)threadIdx.x) >> 1) * 1024)) + ((((int)threadIdx.x) & 1) * 4)));
  B_local[(((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) - ((((int)threadIdx.y) >> 1) * 32))] = *(float4*)(B + (((((((int)threadIdx.y) >> 1) * 1024) + (((int)blockIdx.y) * 128)) + ((((int)threadIdx.y) & 1) * 64)) + (((int)threadIdx.x) * 4)));
    int4 _1 = make_int4(((((((((int)threadIdx.x) & 1) * 512) + ((((int)threadIdx.x) >> 3) * 64)) + (((int)threadIdx.y) * 4)) + ((((int)threadIdx.x) & 7) >> 1)))+(128*0), ((((((((int)threadIdx.x) & 1) * 512) + ((((int)threadIdx.x) >> 3) * 64)) + (((int)threadIdx.y) * 4)) + ((((int)threadIdx.x) & 7) >> 1)))+(128*1), ((((((((int)threadIdx.x) & 1) * 512) + ((((int)threadIdx.x) >> 3) * 64)) + (((int)threadIdx.y) * 4)) + ((((int)threadIdx.x) & 7) >> 1)))+(128*2), ((((((((int)threadIdx.x) & 1) * 512) + ((((int)threadIdx.x) >> 3) * 64)) + (((int)threadIdx.y) * 4)) + ((((int)threadIdx.x) & 7) >> 1)))+(128*3));
    float4 _2 = A_local[0];
    A_shared[_1.x] = _2.x;
    A_shared[_1.y] = _2.y;
    A_shared[_1.z] = _2.z;
    A_shared[_1.w] = _2.w;
  *(float4*)(B_shared + (((((((int)threadIdx.y) >> 1) * 128) + ((((int)threadIdx.x) & 1) * 64)) + ((((int)threadIdx.y) & 1) * 32)) + ((((int)threadIdx.x) >> 1) * 4))) = B_local[(((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) - ((((int)threadIdx.y) >> 1) * 32))];
  for (int i2_0 = 0; i2_0 < 127; ++i2_0) {
    A_local[(((((i2_0 * 8) + ((((int)threadIdx.x) & 1) * 4)) + 8) - max(0, ((((i2_0 * 8) + (((int)threadIdx.x) * 4)) + 8) - ((((int)threadIdx.x) >> 1) * 8)))) / 4)] = *(float4*)(A + ((((((((int)blockIdx.x) * 131072) + (((int)threadIdx.y) * 8192)) + ((((int)threadIdx.x) >> 1) * 1024)) + (i2_0 * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 8));
    B_local[(((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) - ((((int)threadIdx.y) >> 1) * 32))] = *(float4*)(B + ((((((i2_0 * 8192) + ((((int)threadIdx.y) >> 1) * 1024)) + (((int)blockIdx.y) * 128)) + ((((int)threadIdx.y) & 1) * 64)) + (((int)threadIdx.x) * 4)) + 8192));
    __syncthreads();
    *(float4*)(B_shared_local + 0) = *(float4*)(B_shared + (((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)));
    *(float4*)(B_shared_local + 4) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 64));
    *(float4*)(A_shared_local + 0) = *(float4*)(A_shared + (((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)));
    *(float4*)(A_shared_local + 4) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 64));
    for (int i0_2 = 0; i0_2 < 8; ++i0_2) {
      for (int i1_2 = 0; i1_2 < 8; ++i1_2) {
        Y_local[((i0_2 * 8) + i1_2)] = (Y_local[((i0_2 * 8) + i1_2)] + (A_shared_local[i0_2] * B_shared_local[i1_2]));
      }
    }
    *(float4*)(B_shared_local + 0) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 128));
    *(float4*)(B_shared_local + 4) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 192));
    *(float4*)(A_shared_local + 0) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 128));
    *(float4*)(A_shared_local + 4) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 192));
    for (int i0_21 = 0; i0_21 < 8; ++i0_21) {
      for (int i1_21 = 0; i1_21 < 8; ++i1_21) {
        Y_local[((i0_21 * 8) + i1_21)] = (Y_local[((i0_21 * 8) + i1_21)] + (A_shared_local[i0_21] * B_shared_local[i1_21]));
      }
    }
    *(float4*)(B_shared_local + 0) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 256));
    *(float4*)(B_shared_local + 4) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 320));
    *(float4*)(A_shared_local + 0) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 256));
    *(float4*)(A_shared_local + 4) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 320));
    for (int i0_22 = 0; i0_22 < 8; ++i0_22) {
      for (int i1_22 = 0; i1_22 < 8; ++i1_22) {
        Y_local[((i0_22 * 8) + i1_22)] = (Y_local[((i0_22 * 8) + i1_22)] + (A_shared_local[i0_22] * B_shared_local[i1_22]));
      }
    }
    *(float4*)(B_shared_local + 0) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 384));
    *(float4*)(B_shared_local + 4) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 448));
    *(float4*)(A_shared_local + 0) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 384));
    *(float4*)(A_shared_local + 4) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 448));
    for (int i0_23 = 0; i0_23 < 8; ++i0_23) {
      for (int i1_23 = 0; i1_23 < 8; ++i1_23) {
        Y_local[((i0_23 * 8) + i1_23)] = (Y_local[((i0_23 * 8) + i1_23)] + (A_shared_local[i0_23] * B_shared_local[i1_23]));
      }
    }
    *(float4*)(B_shared_local + 0) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 512));
    *(float4*)(B_shared_local + 4) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 576));
    *(float4*)(A_shared_local + 0) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 512));
    *(float4*)(A_shared_local + 4) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 576));
    for (int i0_24 = 0; i0_24 < 8; ++i0_24) {
      for (int i1_24 = 0; i1_24 < 8; ++i1_24) {
        Y_local[((i0_24 * 8) + i1_24)] = (Y_local[((i0_24 * 8) + i1_24)] + (A_shared_local[i0_24] * B_shared_local[i1_24]));
      }
    }
    *(float4*)(B_shared_local + 0) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 640));
    *(float4*)(B_shared_local + 4) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 704));
    *(float4*)(A_shared_local + 0) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 640));
    *(float4*)(A_shared_local + 4) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 704));
    for (int i0_25 = 0; i0_25 < 8; ++i0_25) {
      for (int i1_25 = 0; i1_25 < 8; ++i1_25) {
        Y_local[((i0_25 * 8) + i1_25)] = (Y_local[((i0_25 * 8) + i1_25)] + (A_shared_local[i0_25] * B_shared_local[i1_25]));
      }
    }
    *(float4*)(B_shared_local + 0) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 768));
    *(float4*)(B_shared_local + 4) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 832));
    *(float4*)(A_shared_local + 0) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 768));
    *(float4*)(A_shared_local + 4) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 832));
    for (int i0_26 = 0; i0_26 < 8; ++i0_26) {
      for (int i1_26 = 0; i1_26 < 8; ++i1_26) {
        Y_local[((i0_26 * 8) + i1_26)] = (Y_local[((i0_26 * 8) + i1_26)] + (A_shared_local[i0_26] * B_shared_local[i1_26]));
      }
    }
    *(float4*)(B_shared_local + 0) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 896));
    *(float4*)(B_shared_local + 4) = *(float4*)(B_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.y) * 4)) + 960));
    *(float4*)(A_shared_local + 0) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 896));
    *(float4*)(A_shared_local + 4) = *(float4*)(A_shared + ((((i2_0 & 1) * 1024) + (((int)threadIdx.x) * 4)) + 960));
    for (int i0_27 = 0; i0_27 < 8; ++i0_27) {
      for (int i1_27 = 0; i1_27 < 8; ++i1_27) {
        Y_local[((i0_27 * 8) + i1_27)] = (Y_local[((i0_27 * 8) + i1_27)] + (A_shared_local[i0_27] * B_shared_local[i1_27]));
      }
    }
    __syncthreads();
      int4 _3 = make_int4(((((((((i2_0 + 1) & 1) * 1024) + ((((int)threadIdx.x) & 1) * 512)) + ((((int)threadIdx.x) >> 3) * 64)) + (((int)threadIdx.y) * 4)) + ((((int)threadIdx.x) & 7) >> 1)))+(128*0), ((((((((i2_0 + 1) & 1) * 1024) + ((((int)threadIdx.x) & 1) * 512)) + ((((int)threadIdx.x) >> 3) * 64)) + (((int)threadIdx.y) * 4)) + ((((int)threadIdx.x) & 7) >> 1)))+(128*1), ((((((((i2_0 + 1) & 1) * 1024) + ((((int)threadIdx.x) & 1) * 512)) + ((((int)threadIdx.x) >> 3) * 64)) + (((int)threadIdx.y) * 4)) + ((((int)threadIdx.x) & 7) >> 1)))+(128*2), ((((((((i2_0 + 1) & 1) * 1024) + ((((int)threadIdx.x) & 1) * 512)) + ((((int)threadIdx.x) >> 3) * 64)) + (((int)threadIdx.y) * 4)) + ((((int)threadIdx.x) & 7) >> 1)))+(128*3));
      float4 _4 = A_local[(((((i2_0 * 8) + ((((int)threadIdx.x) & 1) * 4)) + 8) - max(0, ((((i2_0 * 8) + (((int)threadIdx.x) * 4)) + 8) - ((((int)threadIdx.x) >> 1) * 8)))) / 4)];
      A_shared[_3.x] = _4.x;
      A_shared[_3.y] = _4.y;
      A_shared[_3.z] = _4.z;
      A_shared[_3.w] = _4.w;
    *(float4*)(B_shared + (((((((i2_0 + 1) & 1) * 1024) + ((((int)threadIdx.y) >> 1) * 128)) + ((((int)threadIdx.x) & 1) * 64)) + ((((int)threadIdx.y) & 1) * 32)) + ((((int)threadIdx.x) >> 1) * 4))) = B_local[(((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) - ((((int)threadIdx.y) >> 1) * 32))];
  }
  __syncthreads();
  *(float4*)(B_shared_local1 + 0) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1024));
  *(float4*)(B_shared_local1 + 4) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1088));
  *(float4*)(A_shared_local1 + 0) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1024));
  *(float4*)(A_shared_local1 + 4) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1088));
  for (int i0_28 = 0; i0_28 < 8; ++i0_28) {
    for (int i1_28 = 0; i1_28 < 8; ++i1_28) {
      Y_local[((i0_28 * 8) + i1_28)] = (Y_local[((i0_28 * 8) + i1_28)] + (A_shared_local1[i0_28] * B_shared_local1[i1_28]));
    }
  }
  *(float4*)(B_shared_local1 + 0) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1152));
  *(float4*)(B_shared_local1 + 4) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1216));
  *(float4*)(A_shared_local1 + 0) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1152));
  *(float4*)(A_shared_local1 + 4) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1216));
  for (int i0_29 = 0; i0_29 < 8; ++i0_29) {
    for (int i1_29 = 0; i1_29 < 8; ++i1_29) {
      Y_local[((i0_29 * 8) + i1_29)] = (Y_local[((i0_29 * 8) + i1_29)] + (A_shared_local1[i0_29] * B_shared_local1[i1_29]));
    }
  }
  *(float4*)(B_shared_local1 + 0) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1280));
  *(float4*)(B_shared_local1 + 4) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1344));
  *(float4*)(A_shared_local1 + 0) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1280));
  *(float4*)(A_shared_local1 + 4) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1344));
  for (int i0_210 = 0; i0_210 < 8; ++i0_210) {
    for (int i1_210 = 0; i1_210 < 8; ++i1_210) {
      Y_local[((i0_210 * 8) + i1_210)] = (Y_local[((i0_210 * 8) + i1_210)] + (A_shared_local1[i0_210] * B_shared_local1[i1_210]));
    }
  }
  *(float4*)(B_shared_local1 + 0) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1408));
  *(float4*)(B_shared_local1 + 4) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1472));
  *(float4*)(A_shared_local1 + 0) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1408));
  *(float4*)(A_shared_local1 + 4) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1472));
  for (int i0_211 = 0; i0_211 < 8; ++i0_211) {
    for (int i1_211 = 0; i1_211 < 8; ++i1_211) {
      Y_local[((i0_211 * 8) + i1_211)] = (Y_local[((i0_211 * 8) + i1_211)] + (A_shared_local1[i0_211] * B_shared_local1[i1_211]));
    }
  }
  *(float4*)(B_shared_local1 + 0) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1536));
  *(float4*)(B_shared_local1 + 4) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1600));
  *(float4*)(A_shared_local1 + 0) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1536));
  *(float4*)(A_shared_local1 + 4) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1600));
  for (int i0_212 = 0; i0_212 < 8; ++i0_212) {
    for (int i1_212 = 0; i1_212 < 8; ++i1_212) {
      Y_local[((i0_212 * 8) + i1_212)] = (Y_local[((i0_212 * 8) + i1_212)] + (A_shared_local1[i0_212] * B_shared_local1[i1_212]));
    }
  }
  *(float4*)(B_shared_local1 + 0) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1664));
  *(float4*)(B_shared_local1 + 4) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1728));
  *(float4*)(A_shared_local1 + 0) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1664));
  *(float4*)(A_shared_local1 + 4) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1728));
  for (int i0_213 = 0; i0_213 < 8; ++i0_213) {
    for (int i1_213 = 0; i1_213 < 8; ++i1_213) {
      Y_local[((i0_213 * 8) + i1_213)] = (Y_local[((i0_213 * 8) + i1_213)] + (A_shared_local1[i0_213] * B_shared_local1[i1_213]));
    }
  }
  *(float4*)(B_shared_local1 + 0) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1792));
  *(float4*)(B_shared_local1 + 4) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1856));
  *(float4*)(A_shared_local1 + 0) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1792));
  *(float4*)(A_shared_local1 + 4) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1856));
  for (int i0_214 = 0; i0_214 < 8; ++i0_214) {
    for (int i1_214 = 0; i1_214 < 8; ++i1_214) {
      Y_local[((i0_214 * 8) + i1_214)] = (Y_local[((i0_214 * 8) + i1_214)] + (A_shared_local1[i0_214] * B_shared_local1[i1_214]));
    }
  }
  *(float4*)(B_shared_local1 + 0) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1920));
  *(float4*)(B_shared_local1 + 4) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 1984));
  *(float4*)(A_shared_local1 + 0) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1920));
  *(float4*)(A_shared_local1 + 4) = *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1984));
  for (int i0_215 = 0; i0_215 < 8; ++i0_215) {
    for (int i1_215 = 0; i1_215 < 8; ++i1_215) {
      Y_local[((i0_215 * 8) + i1_215)] = (Y_local[((i0_215 * 8) + i1_215)] + (A_shared_local1[i0_215] * B_shared_local1[i1_215]));
    }
  }
  for (int ax0 = 0; ax0 < 8; ++ax0) {
    *(ulonglong4*)(Y + (((((((int)blockIdx.x) * 131072) + (((int)threadIdx.x) * 8192)) + (ax0 * 1024)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 8))) = *(ulonglong4*)(Y_local + (ax0 * 8));
  }
}


