#include "hip/hip_runtime.h"
#include "../include/gemm_header.h"



int main(){

    float flops ;
    int round = 100 ;

    // test cutlass
    cout << "<--------------cutlass--------------->" << endl ;

    flops = get_Gflops(round,run_cutlass) ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<-----------------tvm---------------->" << endl ;

    flops = get_Gflops(round,run_tvm) ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;


    cout << "<----------------v1gemm-------------->" << endl ;

    flops = get_Gflops(round,run_v1gemm) ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<----------------v2gemm-------------->" << endl ;

    flops = get_Gflops(round,run_v2gemm) ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<-----------------end---------------->" << endl ;

    // test err

    cout << "max error: " << get_max_error(run_v1gemm) << endl ; 
    // get_max_error(run_cutlass) ;

}