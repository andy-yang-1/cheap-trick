#include "hip/hip_runtime.h"
#include "../include/gemm_header.h"



int main(){

    float flops , max_err ;
    int round = 100 ;

    // test cutlass
    cout << "<--------------cutlass default--------------->" << endl ;

    flops = get_Gflops(round,run_cutlass) ;
    max_err = get_max_error(run_cutlass) ;
    // cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<-------------cutlass bestPerf-------------->" << endl ;

    flops = get_Gflops(round,run_bestPerf) ;
    max_err = get_max_error(run_bestPerf) ;
    // cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<---------------------tvm-------------------->" << endl ;

    flops = get_Gflops(round,run_tvm) ;
    max_err = get_max_error(run_tvm) ;
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;


    cout << "<--------------------v1gemm------------------>" << endl ;

    flops = get_Gflops(round,run_v1gemm) ;
    max_err = get_max_error(run_v1gemm) ;
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<--------------------v2gemm------------------>" << endl ;

    flops = get_Gflops(round,run_v2gemm) ;
    max_err = get_max_error(run_v2gemm) ;
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<-----------------end---------------->" << endl ;

    // test err

    // cout << "max error: " << get_max_error(run_v1gemm) << endl ; 
    // get_max_error(run_cutlass) ;

}