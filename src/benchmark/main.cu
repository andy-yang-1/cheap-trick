#include "hip/hip_runtime.h"
#include "../include/gemm_header.h"



int main(){

    float flops , max_err ;
    int round = 100 ;

    // test cutlass
    cout << "<--------------cutlass default--------------->" << endl ;

    flops = get_Gflops(round,run_cutlass) ;
    max_err = get_max_error(run_cutlass,true) ;
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<-------------cutlass bestPerf-------------->" << endl ;

    flops = get_Gflops(round,run_bestPerf) ;
    max_err = get_max_error(run_bestPerf,false) ;
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<---------------------tvm-------------------->" << endl ;

    flops = get_Gflops(round,run_tvm) ;
    max_err = get_max_error(run_tvm,true) ;
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;


    cout << "<--------------------v1gemm------------------>" << endl ;

    flops = get_Gflops(round,run_v1gemm) ;
    max_err = get_max_error(run_v1gemm,true) ;
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<--------------------v2gemm------------------>" << endl ;

    flops = get_Gflops(round,run_v2gemm) ;
    max_err = get_max_error(run_v2gemm,true) ;
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<--------------------v3gemm------------------>" << endl ;

    flops = get_Gflops(round,run_v3gemm) ;
    max_err = get_max_error(run_v3gemm,true) ;
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<--------------------v4gemm------------------>" << endl ;

    flops = get_Gflops(round,run_v4gemm) ;
    max_err = get_max_error(run_v4gemm,true) ;
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<-----------------end---------------->" << endl ;

    // test err

    // cout << "max error: " << get_max_error(run_v1gemm) << endl ; 
    // get_max_error(run_cutlass) ;

}