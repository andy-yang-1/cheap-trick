#include "hip/hip_runtime.h"
#include "../include/gemm_header.h"
#include "../include/genGemm.h"

int main(){

    // argv: M N K

    float flops , max_err ;
    int round = 1 ;

    int M = 1009 , N = 1009 , K = 1009 ;
    // int M = 1024 , N = 1024 , K = 1024 ;

    cout << "M: " << M << endl << "N: " << N << endl << "K: " << K << endl ;

    float alpha = 1.0 , beta = 0;


    cout << "<-------------cutlass bestPerf-------------->" << endl ;

    max_err = get_dynamic_max_error(M,N,K,alpha,beta,run_cutlass,true);
    flops = get_dynamic_Gflops(round,M,N,K,alpha,beta,run_cutlass);
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<--------------------genGemm------------------>" << endl ;

    max_err = get_dynamic_max_error(M,N,K,alpha,beta,run_paddinggemm,true);
    flops = get_dynamic_Gflops(round,M,N,K,alpha,beta,run_paddinggemm);
    // max_err = get_dynamic_max_error(M,N,K,alpha,beta,run_v4gemm,true);
    // flops = get_dynamic_Gflops(round,M,N,K,alpha,beta,run_v4gemm);
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<-----------------end---------------->" << endl ;

}