#include "hip/hip_runtime.h"
#include "../include/gemm_header.h"
#include "../include/genGemm.h"

int main(){

    // argv: M N K

    float flops , max_err ;
    int round = 1 ;

    int M = 1000 , N = 2000 , K = 3000 ;
    // int M = 2000 , N = 2024 , K = 2024 ;

    cout << "M: " << M << endl << "N: " << N << endl << "K: " << K << endl ;

    float alpha = 1.0 , beta = 0;


    cout << "<-------------cutlass bestPerf-------------->" << endl ;

    max_err = get_dynamic_max_error(M,N,K,alpha,beta,run_cutlass,true);
    flops = get_dynamic_Gflops(round,M,N,K,alpha,beta,run_cutlass);
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<--------------------genGemm------------------>" << endl ;

    max_err = get_dynamic_max_error(M,N,K,alpha,beta,run_genGemm,true);
    flops = get_dynamic_Gflops(round,M,N,K,alpha,beta,run_genGemm);
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<-----------------end---------------->" << endl ;

}