#include "hip/hip_runtime.h"
#include "../include/gemm_header.h"
#include "../include/cutlassMultiStage.h"

int main(){
    // // cout << "hello world" << endl ;
    // // cout << "max error: " << get_max_error(run_v4gemm,false) << endl  ;
    // cout << "max error: " << get_max_error(run_v4gemm,true) << endl  ;
    cout << "gflops: " << get_Gflops(100,run_v5gemm) << endl  ;
    cout << "error rate: " << get_max_error(run_v5gemm,true) << endl ;

    // cout << "gflops: " << get_Gflops(100,run_cutlassMultiStage) << endl  ;
    // cout << "error rate: " << get_max_error(run_cutlassMultiStage,false) << endl ;
    // cout << "max error: " << get_max_error(run_v4gemm,true) << endl  ;

    // int M = 1024;           //M
    // int N = 1024;           //N
    // int K = 1024;           //K

    // float alpha = 10;      //alpha
    // float beta = 10.0;       //beta

    // float *A;               //申明A矩阵host端指针
    // float *B;               //申明B矩阵host端指针
    // float *C;               //申明C矩阵host端指针
    // float *D;               //申明D矩阵host端指针


    // size_t A_mem_size = sizeof(float) * M * K; //memory size of matrix A = M * K * sizeof(float)
    // size_t B_mem_size = sizeof(float) * K * N; //memory size of matrix B = K * N * sizeof(float)
    // size_t C_mem_size = sizeof(float) * M * N; //memory size of matrix C = M * N * sizeof(float)
    // size_t D_mem_size = sizeof(float) * M * N; //memory size of matrix C = M * N * sizeof(float)
 
    // A = (float*)malloc(A_mem_size);  // host端A矩阵分配内存
    // B = (float*)malloc(B_mem_size);  // host端B矩阵分配内存
    // C = (float*)malloc(C_mem_size);  // host端C矩阵分配内存
    // D = (float*)malloc(D_mem_size);  // host端D矩阵分配内存

    // float *d_A;            // 申明device端A矩阵的指针
    // float *d_B;            // 申明device端B矩阵的指针
    // float *d_C;            // 申明device端C矩阵的指针
    // float *d_D;            // 申明device端D矩阵的指针
    // // float *d_zero ;          // 申明device端全0矩阵

    // hipMalloc((void**)&d_A, A_mem_size);  // device端为A矩阵分配内存
    // hipMalloc((void**)&d_B, B_mem_size);  // device端为B矩阵分配内存
    // hipMalloc((void**)&d_C, C_mem_size);  // device端为C矩阵分配内存
    // hipMalloc((void**)&d_D, D_mem_size);  // device端为D矩阵分配内存

    // // generate_tensor_2D(A, M, K);     // 填充A矩阵
    // // generate_tensor_2D(B, K, N);     // 填充B矩阵  
    // generate_const_2D(A,M,K,1) ;
    // generate_const_2D(B,K,N,1) ;

    // hipMemcpy(d_A, A, A_mem_size, hipMemcpyHostToDevice); // 将矩阵A的数据传递到device端
    // hipMemcpy(d_B, B, B_mem_size, hipMemcpyHostToDevice); // 将矩阵B的数据传递到device端
    // // hipMemcpy(d_zero, zero, D_mem_size, hipMemcpyHostToDevice); // 将矩阵C的数据传递到device端

    // // v4 change A & B

   
    // run_cutlass(M,N,K,d_A,d_B,d_D,alpha,beta) ;
    // run_v4gemm(M,N,K,d_A,d_B,d_C,alpha,beta) ;


    // hipMemcpy(C,d_C,C_mem_size,hipMemcpyDeviceToHost) ;
    // hipMemcpy(D,d_D,D_mem_size,hipMemcpyDeviceToHost) ;


    // float max_err = 0 ;

    // for (int i = 0 ; i < M * N ; i++){
    //     if (abs(C[i]-D[i])>3){
    //         // cout << "i: " << i << " C[i] = " << C[i] << endl ;
    //         // cout << "i: " << i << " D[i] = " << D[i] << endl ;
            
    //         // break ;
    //     }
    //     max_err = max_err > abs(C[i]-D[i]) ? max_err : abs(C[i]-D[i]) ;
            
    // }

    // cout << "max error: " << max_err << endl ;
    // cout << "v4: " <<endl ;
    // cout << C[0] << endl << C[1024] << endl ;
    // cout << "cutlass: " << endl ;
    // cout << D[0] << endl << D[1024] << endl ;

}