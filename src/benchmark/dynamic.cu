#include "hip/hip_runtime.h"
#include "../include/gemm_header.h"
#include "../include/genGemm.h"

int main(int argc,char* argv[]){

    // argv: M N K

    float flops , max_err ;
    int round = 100 ;

    int M = atoi(argv[1]) , N = atoi(argv[2]) , K = atoi(argv[3]) ;

    cout << "M: " << M << endl << "N: " << N << endl << "K: " << K << endl ;

    float alpha = 1.0 , beta = 0;


    cout << "<-------------cutlass bestPerf-------------->" << endl ;

    max_err = get_dynamic_max_error(M,N,K,alpha,beta,run_cutlass,true);
    flops = get_dynamic_Gflops(round,M,N,K,alpha,beta,run_cutlass);
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<--------------------genGemm------------------>" << endl ;

    max_err = get_dynamic_max_error(M,N,K,alpha,beta,run_genGemm,true);
    flops = get_dynamic_Gflops(round,M,N,K,alpha,beta,run_genGemm);
    cout << "max error: " << max_err << endl ;
    cout << "rounds: " << round << endl ;
    cout << "average gflops: " << flops << endl ;

    cout << "<-----------------end---------------->" << endl ;

}