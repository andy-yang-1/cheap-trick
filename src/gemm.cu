
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mm_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[64];
  float2 A_local[1];
  ulonglong4 B_local[16];
  __shared__ float A_shared[256];
  __shared__ float B_shared[1024];
  float A_shared_local[16];
  float B_shared_local[4];
  float A_shared_local_1[16];
  float B_shared_local_1[4];
  for (int i0_2_init = 0; i0_2_init < 16; ++i0_2_init) {
    for (int i1_2_init = 0; i1_2_init < 4; ++i1_2_init) {
      Y_local[((i0_2_init * 4) + i1_2_init)] = 0.000000e+00f;
    }
  }
  A_local[0] = *(float2*)(A + (((((int)blockIdx.x) * 36864) + (((int)threadIdx.y) * 1152)) + (((int)threadIdx.x) * 2)));
  B_local[(((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) - ((((int)threadIdx.y) >> 3) * 16))] = *(ulonglong4*)(B + (((((((int)threadIdx.y) >> 3) * 1152) + (((int)blockIdx.y) * 128)) + ((((int)threadIdx.y) & 7) * 16)) + (((int)threadIdx.x) * 8)));
    int2 __1 = make_int2((((((int)threadIdx.x) * 64) + ((int)threadIdx.y)))+(32*0), (((((int)threadIdx.x) * 64) + ((int)threadIdx.y)))+(32*1));
    float2 __2 = A_local[0];
    A_shared[__1.x] = __2.x;
    A_shared[__1.y] = __2.y;
  *(ulonglong4*)(B_shared + ((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 8))) = B_local[(((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) - ((((int)threadIdx.y) >> 3) * 16))];
  for (int i2_0 = 0; i2_0 < 287; ++i2_0) {
    A_local[0] = *(float2*)(A + (((((((int)blockIdx.x) * 36864) + (((int)threadIdx.y) * 1152)) + (i2_0 * 4)) + (((int)threadIdx.x) * 2)) + 4));
    B_local[(((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) - ((((int)threadIdx.y) >> 3) * 16))] = *(ulonglong4*)(B + ((((((i2_0 * 4608) + ((((int)threadIdx.y) >> 3) * 1152)) + (((int)blockIdx.y) * 128)) + ((((int)threadIdx.y) & 7) * 16)) + (((int)threadIdx.x) * 8)) + 4608));
    __syncthreads();
    A_shared_local[0] = A_shared[(((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16))];
    A_shared_local[1] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 1)];
    A_shared_local[2] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 2)];
    A_shared_local[3] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 3)];
    A_shared_local[4] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 4)];
    A_shared_local[5] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 5)];
    A_shared_local[6] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 6)];
    A_shared_local[7] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 7)];
    A_shared_local[8] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 8)];
    A_shared_local[9] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 9)];
    A_shared_local[10] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 10)];
    A_shared_local[11] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 11)];
    A_shared_local[12] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 12)];
    A_shared_local[13] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 13)];
    A_shared_local[14] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 14)];
    A_shared_local[15] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 15)];
    *(float4*)(B_shared_local + 0) = *(float4*)(B_shared + (((i2_0 & 1) * 512) + (((int)threadIdx.y) * 4)));
    for (int i0_2 = 0; i0_2 < 16; ++i0_2) {
      for (int i1_2 = 0; i1_2 < 4; ++i1_2) {
        Y_local[((i0_2 * 4) + i1_2)] = (Y_local[((i0_2 * 4) + i1_2)] + (A_shared_local[i0_2] * B_shared_local[i1_2]));
      }
    }
    A_shared_local[0] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 32)];
    A_shared_local[1] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 33)];
    A_shared_local[2] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 34)];
    A_shared_local[3] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 35)];
    A_shared_local[4] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 36)];
    A_shared_local[5] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 37)];
    A_shared_local[6] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 38)];
    A_shared_local[7] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 39)];
    A_shared_local[8] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 40)];
    A_shared_local[9] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 41)];
    A_shared_local[10] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 42)];
    A_shared_local[11] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 43)];
    A_shared_local[12] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 44)];
    A_shared_local[13] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 45)];
    A_shared_local[14] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 46)];
    A_shared_local[15] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 47)];
    *(float4*)(B_shared_local + 0) = *(float4*)(B_shared + ((((i2_0 & 1) * 512) + (((int)threadIdx.y) * 4)) + 128));
    for (int i0_2_1 = 0; i0_2_1 < 16; ++i0_2_1) {
      for (int i1_2_1 = 0; i1_2_1 < 4; ++i1_2_1) {
        Y_local[((i0_2_1 * 4) + i1_2_1)] = (Y_local[((i0_2_1 * 4) + i1_2_1)] + (A_shared_local[i0_2_1] * B_shared_local[i1_2_1]));
      }
    }
    A_shared_local[0] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 64)];
    A_shared_local[1] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 65)];
    A_shared_local[2] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 66)];
    A_shared_local[3] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 67)];
    A_shared_local[4] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 68)];
    A_shared_local[5] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 69)];
    A_shared_local[6] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 70)];
    A_shared_local[7] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 71)];
    A_shared_local[8] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 72)];
    A_shared_local[9] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 73)];
    A_shared_local[10] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 74)];
    A_shared_local[11] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 75)];
    A_shared_local[12] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 76)];
    A_shared_local[13] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 77)];
    A_shared_local[14] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 78)];
    A_shared_local[15] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 79)];
    *(float4*)(B_shared_local + 0) = *(float4*)(B_shared + ((((i2_0 & 1) * 512) + (((int)threadIdx.y) * 4)) + 256));
    for (int i0_2_2 = 0; i0_2_2 < 16; ++i0_2_2) {
      for (int i1_2_2 = 0; i1_2_2 < 4; ++i1_2_2) {
        Y_local[((i0_2_2 * 4) + i1_2_2)] = (Y_local[((i0_2_2 * 4) + i1_2_2)] + (A_shared_local[i0_2_2] * B_shared_local[i1_2_2]));
      }
    }
    A_shared_local[0] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 96)];
    A_shared_local[1] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 97)];
    A_shared_local[2] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 98)];
    A_shared_local[3] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 99)];
    A_shared_local[4] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 100)];
    A_shared_local[5] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 101)];
    A_shared_local[6] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 102)];
    A_shared_local[7] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 103)];
    A_shared_local[8] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 104)];
    A_shared_local[9] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 105)];
    A_shared_local[10] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 106)];
    A_shared_local[11] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 107)];
    A_shared_local[12] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 108)];
    A_shared_local[13] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 109)];
    A_shared_local[14] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 110)];
    A_shared_local[15] = A_shared[((((i2_0 & 1) * 128) + (((int)threadIdx.x) * 16)) + 111)];
    *(float4*)(B_shared_local + 0) = *(float4*)(B_shared + ((((i2_0 & 1) * 512) + (((int)threadIdx.y) * 4)) + 384));
    for (int i0_2_3 = 0; i0_2_3 < 16; ++i0_2_3) {
      for (int i1_2_3 = 0; i1_2_3 < 4; ++i1_2_3) {
        Y_local[((i0_2_3 * 4) + i1_2_3)] = (Y_local[((i0_2_3 * 4) + i1_2_3)] + (A_shared_local[i0_2_3] * B_shared_local[i1_2_3]));
      }
    }
    __syncthreads();
      int2 __3 = make_int2(((((((i2_0 + 1) & 1) * 128) + (((int)threadIdx.x) * 64)) + ((int)threadIdx.y)))+(32*0), ((((((i2_0 + 1) & 1) * 128) + (((int)threadIdx.x) * 64)) + ((int)threadIdx.y)))+(32*1));
      float2 __4 = A_local[0];
      A_shared[__3.x] = __4.x;
      A_shared[__3.y] = __4.y;
    *(ulonglong4*)(B_shared + (((((i2_0 + 1) & 1) * 512) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 8))) = B_local[(((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) - ((((int)threadIdx.y) >> 3) * 16))];
  }
  __syncthreads();
  A_shared_local_1[0] = A_shared[((((int)threadIdx.x) * 16) + 128)];
  A_shared_local_1[1] = A_shared[((((int)threadIdx.x) * 16) + 129)];
  A_shared_local_1[2] = A_shared[((((int)threadIdx.x) * 16) + 130)];
  A_shared_local_1[3] = A_shared[((((int)threadIdx.x) * 16) + 131)];
  A_shared_local_1[4] = A_shared[((((int)threadIdx.x) * 16) + 132)];
  A_shared_local_1[5] = A_shared[((((int)threadIdx.x) * 16) + 133)];
  A_shared_local_1[6] = A_shared[((((int)threadIdx.x) * 16) + 134)];
  A_shared_local_1[7] = A_shared[((((int)threadIdx.x) * 16) + 135)];
  A_shared_local_1[8] = A_shared[((((int)threadIdx.x) * 16) + 136)];
  A_shared_local_1[9] = A_shared[((((int)threadIdx.x) * 16) + 137)];
  A_shared_local_1[10] = A_shared[((((int)threadIdx.x) * 16) + 138)];
  A_shared_local_1[11] = A_shared[((((int)threadIdx.x) * 16) + 139)];
  A_shared_local_1[12] = A_shared[((((int)threadIdx.x) * 16) + 140)];
  A_shared_local_1[13] = A_shared[((((int)threadIdx.x) * 16) + 141)];
  A_shared_local_1[14] = A_shared[((((int)threadIdx.x) * 16) + 142)];
  A_shared_local_1[15] = A_shared[((((int)threadIdx.x) * 16) + 143)];
  *(float4*)(B_shared_local_1 + 0) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 512));
  for (int i0_2_4 = 0; i0_2_4 < 16; ++i0_2_4) {
    for (int i1_2_4 = 0; i1_2_4 < 4; ++i1_2_4) {
      Y_local[((i0_2_4 * 4) + i1_2_4)] = (Y_local[((i0_2_4 * 4) + i1_2_4)] + (A_shared_local_1[i0_2_4] * B_shared_local_1[i1_2_4]));
    }
  }
  A_shared_local_1[0] = A_shared[((((int)threadIdx.x) * 16) + 160)];
  A_shared_local_1[1] = A_shared[((((int)threadIdx.x) * 16) + 161)];
  A_shared_local_1[2] = A_shared[((((int)threadIdx.x) * 16) + 162)];
  A_shared_local_1[3] = A_shared[((((int)threadIdx.x) * 16) + 163)];
  A_shared_local_1[4] = A_shared[((((int)threadIdx.x) * 16) + 164)];
  A_shared_local_1[5] = A_shared[((((int)threadIdx.x) * 16) + 165)];
  A_shared_local_1[6] = A_shared[((((int)threadIdx.x) * 16) + 166)];
  A_shared_local_1[7] = A_shared[((((int)threadIdx.x) * 16) + 167)];
  A_shared_local_1[8] = A_shared[((((int)threadIdx.x) * 16) + 168)];
  A_shared_local_1[9] = A_shared[((((int)threadIdx.x) * 16) + 169)];
  A_shared_local_1[10] = A_shared[((((int)threadIdx.x) * 16) + 170)];
  A_shared_local_1[11] = A_shared[((((int)threadIdx.x) * 16) + 171)];
  A_shared_local_1[12] = A_shared[((((int)threadIdx.x) * 16) + 172)];
  A_shared_local_1[13] = A_shared[((((int)threadIdx.x) * 16) + 173)];
  A_shared_local_1[14] = A_shared[((((int)threadIdx.x) * 16) + 174)];
  A_shared_local_1[15] = A_shared[((((int)threadIdx.x) * 16) + 175)];
  *(float4*)(B_shared_local_1 + 0) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 640));
  for (int i0_2_5 = 0; i0_2_5 < 16; ++i0_2_5) {
    for (int i1_2_5 = 0; i1_2_5 < 4; ++i1_2_5) {
      Y_local[((i0_2_5 * 4) + i1_2_5)] = (Y_local[((i0_2_5 * 4) + i1_2_5)] + (A_shared_local_1[i0_2_5] * B_shared_local_1[i1_2_5]));
    }
  }
  A_shared_local_1[0] = A_shared[((((int)threadIdx.x) * 16) + 192)];
  A_shared_local_1[1] = A_shared[((((int)threadIdx.x) * 16) + 193)];
  A_shared_local_1[2] = A_shared[((((int)threadIdx.x) * 16) + 194)];
  A_shared_local_1[3] = A_shared[((((int)threadIdx.x) * 16) + 195)];
  A_shared_local_1[4] = A_shared[((((int)threadIdx.x) * 16) + 196)];
  A_shared_local_1[5] = A_shared[((((int)threadIdx.x) * 16) + 197)];
  A_shared_local_1[6] = A_shared[((((int)threadIdx.x) * 16) + 198)];
  A_shared_local_1[7] = A_shared[((((int)threadIdx.x) * 16) + 199)];
  A_shared_local_1[8] = A_shared[((((int)threadIdx.x) * 16) + 200)];
  A_shared_local_1[9] = A_shared[((((int)threadIdx.x) * 16) + 201)];
  A_shared_local_1[10] = A_shared[((((int)threadIdx.x) * 16) + 202)];
  A_shared_local_1[11] = A_shared[((((int)threadIdx.x) * 16) + 203)];
  A_shared_local_1[12] = A_shared[((((int)threadIdx.x) * 16) + 204)];
  A_shared_local_1[13] = A_shared[((((int)threadIdx.x) * 16) + 205)];
  A_shared_local_1[14] = A_shared[((((int)threadIdx.x) * 16) + 206)];
  A_shared_local_1[15] = A_shared[((((int)threadIdx.x) * 16) + 207)];
  *(float4*)(B_shared_local_1 + 0) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 768));
  for (int i0_2_6 = 0; i0_2_6 < 16; ++i0_2_6) {
    for (int i1_2_6 = 0; i1_2_6 < 4; ++i1_2_6) {
      Y_local[((i0_2_6 * 4) + i1_2_6)] = (Y_local[((i0_2_6 * 4) + i1_2_6)] + (A_shared_local_1[i0_2_6] * B_shared_local_1[i1_2_6]));
    }
  }
  A_shared_local_1[0] = A_shared[((((int)threadIdx.x) * 16) + 224)];
  A_shared_local_1[1] = A_shared[((((int)threadIdx.x) * 16) + 225)];
  A_shared_local_1[2] = A_shared[((((int)threadIdx.x) * 16) + 226)];
  A_shared_local_1[3] = A_shared[((((int)threadIdx.x) * 16) + 227)];
  A_shared_local_1[4] = A_shared[((((int)threadIdx.x) * 16) + 228)];
  A_shared_local_1[5] = A_shared[((((int)threadIdx.x) * 16) + 229)];
  A_shared_local_1[6] = A_shared[((((int)threadIdx.x) * 16) + 230)];
  A_shared_local_1[7] = A_shared[((((int)threadIdx.x) * 16) + 231)];
  A_shared_local_1[8] = A_shared[((((int)threadIdx.x) * 16) + 232)];
  A_shared_local_1[9] = A_shared[((((int)threadIdx.x) * 16) + 233)];
  A_shared_local_1[10] = A_shared[((((int)threadIdx.x) * 16) + 234)];
  A_shared_local_1[11] = A_shared[((((int)threadIdx.x) * 16) + 235)];
  A_shared_local_1[12] = A_shared[((((int)threadIdx.x) * 16) + 236)];
  A_shared_local_1[13] = A_shared[((((int)threadIdx.x) * 16) + 237)];
  A_shared_local_1[14] = A_shared[((((int)threadIdx.x) * 16) + 238)];
  A_shared_local_1[15] = A_shared[((((int)threadIdx.x) * 16) + 239)];
  *(float4*)(B_shared_local_1 + 0) = *(float4*)(B_shared + ((((int)threadIdx.y) * 4) + 896));
  for (int i0_2_7 = 0; i0_2_7 < 16; ++i0_2_7) {
    for (int i1_2_7 = 0; i1_2_7 < 4; ++i1_2_7) {
      Y_local[((i0_2_7 * 4) + i1_2_7)] = (Y_local[((i0_2_7 * 4) + i1_2_7)] + (A_shared_local_1[i0_2_7] * B_shared_local_1[i1_2_7]));
    }
  }
  for (int ax0 = 0; ax0 < 16; ++ax0) {
    *(float4*)(Y + (((((((int)blockIdx.x) * 36864) + (((int)threadIdx.x) * 18432)) + (ax0 * 1152)) + (((int)blockIdx.y) * 128)) + (((int)threadIdx.y) * 4))) = *(float4*)(Y_local + (ax0 * 4));
  }
}


