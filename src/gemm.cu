
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mm_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[40];
  float A_local[1];
  float B_local[1];
  __shared__ float A_shared[512];
  __shared__ float B_shared[192];
  float A_shared_local[10];
  float B_shared_local[4];
  float A_shared_local_1[10];
  float B_shared_local_1[4];
  for (int i0_2_init = 0; i0_2_init < 10; ++i0_2_init) {
    for (int i1_2_init = 0; i1_2_init < 4; ++i1_2_init) {
      Y_local[((i0_2_init * 4) + i1_2_init)] = 0.000000e+00f;
    }
  }
  if (((int)threadIdx.y) < 10) {
    A_local[0] = A[(((((int)blockIdx.x) * 26680) + (((int)threadIdx.y) * 2668)) + (((int)threadIdx.x) * 667))];
  }
  B_local[0] = B[(((((int)blockIdx.y) * 64) + (((int)threadIdx.y) * 4)) + ((int)threadIdx.x))];
  if (((int)threadIdx.y) < 10) {
    A_shared[((((((((((int)blockIdx.x) * 5) + (((int)threadIdx.y) >> 1)) >> 4) * 128) + ((((int)threadIdx.y) & 1) * 64)) + ((((((int)blockIdx.x) * 5) + (((int)threadIdx.y) >> 1)) & 15) * 4)) + ((int)threadIdx.x)) - (((((int)blockIdx.x) * 40) >> 7) * 128))] = A_local[0];
  }
  B_shared[((((((int)threadIdx.y) & 1) * 64) + ((((int)threadIdx.y) >> 1) * 4)) + ((int)threadIdx.x))] = B_local[0];
  for (int i2_0 = 0; i2_0 < 666; ++i2_0) {
    if (((int)threadIdx.y) < 10) {
      A_local[0] = A[(((((((int)blockIdx.x) * 26680) + (((int)threadIdx.y) * 2668)) + (((int)threadIdx.x) * 667)) + i2_0) + 1)];
    }
    B_local[0] = B[(((((i2_0 * 1152) + (((int)blockIdx.y) * 64)) + (((int)threadIdx.y) * 4)) + ((int)threadIdx.x)) + 1152)];
    __syncthreads();
    for (int ax0_ax1_fused_1_s = 0; ax0_ax1_fused_1_s < 4; ++ax0_ax1_fused_1_s) {
      A_shared_local[ax0_ax1_fused_1_s] = A_shared[(((((((i2_0 & 1) * 256) + (((((((int)blockIdx.x) * 40) + (((int)threadIdx.x) * 10)) + ax0_ax1_fused_1_s) >> 7) * 128)) + (((((ax0_ax1_fused_1_s >> 1) + ((int)threadIdx.x)) & 3) >> 1) * 64)) + ((((((int)blockIdx.x) * 5) + (((((int)threadIdx.x) * 10) + ax0_ax1_fused_1_s) >> 3)) & 15) * 4)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_1_s) & 3)) - (((((int)blockIdx.x) * 40) >> 7) * 128))];
    }
    for (int ax0_ax1_fused_1_s_1 = 0; ax0_ax1_fused_1_s_1 < 4; ++ax0_ax1_fused_1_s_1) {
      A_shared_local[(ax0_ax1_fused_1_s_1 + 4)] = A_shared[(((((((i2_0 & 1) * 256) + ((((((((int)blockIdx.x) * 40) + (((int)threadIdx.x) * 10)) + ax0_ax1_fused_1_s_1) + 4) >> 7) * 128)) + ((((((((int)threadIdx.x) * 5) + (ax0_ax1_fused_1_s_1 >> 1)) >> 1) + 1) & 1) * 64)) + ((((((int)blockIdx.x) * 5) + ((((((int)threadIdx.x) * 10) + ax0_ax1_fused_1_s_1) + 4) >> 3)) & 15) * 4)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_1_s_1) & 3)) - (((((int)blockIdx.x) * 40) >> 7) * 128))];
    }
    for (int ax0_ax1_fused_1_s_2 = 0; ax0_ax1_fused_1_s_2 < 4; ++ax0_ax1_fused_1_s_2) {
      if (ax0_ax1_fused_1_s_2 < 2) {
        A_shared_local[(ax0_ax1_fused_1_s_2 + 8)] = A_shared[((((((((i2_0 & 1) * 256) + ((((((((int)blockIdx.x) * 40) + (((int)threadIdx.x) * 10)) + ax0_ax1_fused_1_s_2) + 8) >> 7) * 128)) + ((((int)threadIdx.x) >> 1) * 64)) + (((((((int)blockIdx.x) * 5) + ((int)threadIdx.x)) + 1) & 15) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax0_ax1_fused_1_s_2) - (((((int)blockIdx.x) * 40) >> 7) * 128))];
      }
    }
    *(float4*)(B_shared_local + 0) = *(float4*)(B_shared + ((((i2_0 & 1) * 96) + ((((int)threadIdx.y) & 1) * 64)) + ((((int)threadIdx.y) >> 1) * 4)));
    for (int i0_2 = 0; i0_2 < 10; ++i0_2) {
      for (int i1_2 = 0; i1_2 < 4; ++i1_2) {
        Y_local[((i0_2 * 4) + i1_2)] = (Y_local[((i0_2 * 4) + i1_2)] + (A_shared_local[i0_2] * B_shared_local[i1_2]));
      }
    }
    __syncthreads();
    if (((int)threadIdx.y) < 10) {
      A_shared[((((((((i2_0 + 1) & 1) * 256) + ((((((int)blockIdx.x) * 5) + (((int)threadIdx.y) >> 1)) >> 4) * 128)) + ((((int)threadIdx.y) & 1) * 64)) + ((((((int)blockIdx.x) * 5) + (((int)threadIdx.y) >> 1)) & 15) * 4)) + ((int)threadIdx.x)) - (((((int)blockIdx.x) * 40) >> 7) * 128))] = A_local[0];
    }
    B_shared[((((((i2_0 + 1) & 1) * 96) + ((((int)threadIdx.y) & 1) * 64)) + ((((int)threadIdx.y) >> 1) * 4)) + ((int)threadIdx.x))] = B_local[0];
  }
  __syncthreads();
  for (int ax0_ax1_fused_1_s_3 = 0; ax0_ax1_fused_1_s_3 < 4; ++ax0_ax1_fused_1_s_3) {
    A_shared_local_1[ax0_ax1_fused_1_s_3] = A_shared[(((((((((((int)blockIdx.x) * 40) + (((int)threadIdx.x) * 10)) + ax0_ax1_fused_1_s_3) >> 7) * 128) + (((((ax0_ax1_fused_1_s_3 >> 1) + ((int)threadIdx.x)) & 3) >> 1) * 64)) + ((((((int)blockIdx.x) * 5) + (((((int)threadIdx.x) * 10) + ax0_ax1_fused_1_s_3) >> 3)) & 15) * 4)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_1_s_3) & 3)) - (((((int)blockIdx.x) * 40) >> 7) * 128))];
  }
  for (int ax0_ax1_fused_1_s_4 = 0; ax0_ax1_fused_1_s_4 < 4; ++ax0_ax1_fused_1_s_4) {
    A_shared_local_1[(ax0_ax1_fused_1_s_4 + 4)] = A_shared[((((((((((((int)blockIdx.x) * 40) + (((int)threadIdx.x) * 10)) + ax0_ax1_fused_1_s_4) + 4) >> 7) * 128) + ((((((((int)threadIdx.x) * 5) + (ax0_ax1_fused_1_s_4 >> 1)) >> 1) + 1) & 1) * 64)) + ((((((int)blockIdx.x) * 5) + ((((((int)threadIdx.x) * 10) + ax0_ax1_fused_1_s_4) + 4) >> 3)) & 15) * 4)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_1_s_4) & 3)) - (((((int)blockIdx.x) * 40) >> 7) * 128))];
  }
  for (int ax0_ax1_fused_1_s_5 = 0; ax0_ax1_fused_1_s_5 < 4; ++ax0_ax1_fused_1_s_5) {
    if (ax0_ax1_fused_1_s_5 < 2) {
      A_shared_local_1[(ax0_ax1_fused_1_s_5 + 8)] = A_shared[(((((((((((((int)blockIdx.x) * 40) + (((int)threadIdx.x) * 10)) + ax0_ax1_fused_1_s_5) + 8) >> 7) * 128) + ((((int)threadIdx.x) >> 1) * 64)) + (((((((int)blockIdx.x) * 5) + ((int)threadIdx.x)) + 1) & 15) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax0_ax1_fused_1_s_5) - (((((int)blockIdx.x) * 40) >> 7) * 128))];
    }
  }
  *(float4*)(B_shared_local_1 + 0) = *(float4*)(B_shared + (((((int)threadIdx.y) & 1) * 64) + ((((int)threadIdx.y) >> 1) * 4)));
  for (int i0_2_1 = 0; i0_2_1 < 10; ++i0_2_1) {
    for (int i1_2_1 = 0; i1_2_1 < 4; ++i1_2_1) {
      Y_local[((i0_2_1 * 4) + i1_2_1)] = (Y_local[((i0_2_1 * 4) + i1_2_1)] + (A_shared_local_1[i0_2_1] * B_shared_local_1[i1_2_1]));
    }
  }
  for (int ax0 = 0; ax0 < 10; ++ax0) {
    *(float4*)(Y + (((((((int)blockIdx.x) * 46080) + (((int)threadIdx.x) * 11520)) + (ax0 * 1152)) + (((int)blockIdx.y) * 64)) + (((int)threadIdx.y) * 4))) = *(float4*)(Y_local + (ax0 * 4));
  }
}


