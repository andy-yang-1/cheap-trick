
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mm_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[64];
  float A_local[256];
  float2 B_local[1024];
  __shared__ float A_local_shared[1152];
  __shared__ float B_local_shared[4096];
  float A_local_shared_local[1];
  float B_local_shared_local[4];
  float A_local_shared_local_1[1];
  float B_local_shared_local_1[4];
  for (int i0_3_init = 0; i0_3_init < 16; ++i0_3_init) {
    Y_local[i0_3_init] = 0.000000e+00f;
    Y_local[(i0_3_init + 16)] = 0.000000e+00f;
    Y_local[(i0_3_init + 32)] = 0.000000e+00f;
    Y_local[(i0_3_init + 48)] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_3_s = 0; ax0_ax1_fused_3_s < 4; ++ax0_ax1_fused_3_s) {
    if (((int)threadIdx.y) < 32) {
      A_local[((((((int)threadIdx.y) * 4) + ax0_ax1_fused_3_s) - min(((((int)threadIdx.y) & 1) * 4), ((((int)threadIdx.y) & 3) * 2))) - (min((((int)threadIdx.y) >> 1), (((int)threadIdx.y) >> 2)) * 8))] = A[((((((int)blockIdx.x) * 16384) + ((((int)threadIdx.y) >> 1) * 1024)) + ((((int)threadIdx.y) & 1) * 4)) + ax0_ax1_fused_3_s)];
    }
  }
  for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 16; ++ax0_ax1_fused_0) {
    B_local[((ax0_ax1_fused_0 * 64) + ((int)threadIdx.y))] = *(float2*)(B + (((((ax0_ax1_fused_0 >> 1) * 1024) + (((int)blockIdx.y) * 256)) + ((ax0_ax1_fused_0 & 1) * 128)) + (((int)threadIdx.y) * 2)));
  }
    int2 __1 = make_int2(((((((((int)threadIdx.y) & 3) * 144) + (((((int)threadIdx.y) & 31) >> 4) * 64)) + ((((int)threadIdx.y) >> 5) * 4)) + ((((int)threadIdx.y) & 15) >> 2)))+(72*0), ((((((((int)threadIdx.y) & 3) * 144) + (((((int)threadIdx.y) & 31) >> 4) * 64)) + ((((int)threadIdx.y) >> 5) * 4)) + ((((int)threadIdx.y) & 15) >> 2)))+(72*1));
    float2 __2 = *(float2*)(A_local + (((((int)threadIdx.y) * 2) - min(((((int)threadIdx.y) & 1) * 4), ((((int)threadIdx.y) & 3) * 2))) - (min((((int)threadIdx.y) >> 1), (((int)threadIdx.y) >> 2)) * 8)));
    A_local_shared[__1.x] = __2.x;
    A_local_shared[__1.y] = __2.y;
  for (int ax0_ax1_fused_0_1 = 0; ax0_ax1_fused_0_1 < 16; ++ax0_ax1_fused_0_1) {
    *(float2*)(B_local_shared + ((((ax0_ax1_fused_0_1 * 128) + (((((int)threadIdx.y) & 3) >> 1) * 64)) + ((((int)threadIdx.y) >> 2) * 4)) + ((((int)threadIdx.y) & 1) * 2))) = B_local[((ax0_ax1_fused_0_1 * 64) + ((int)threadIdx.y))];
  }
  for (int i2_0 = 0; i2_0 < 127; ++i2_0) {
    for (int ax0_ax1_fused_3_s_1 = 0; ax0_ax1_fused_3_s_1 < 4; ++ax0_ax1_fused_3_s_1) {
      if (((int)threadIdx.y) < 32) {
        A_local[((((((int)threadIdx.y) * 4) + ax0_ax1_fused_3_s_1) - min(((((int)threadIdx.y) & 1) * 4), ((((int)threadIdx.y) & 3) * 2))) - (min((((int)threadIdx.y) >> 1), (((int)threadIdx.y) >> 2)) * 8))] = A[((((((((int)blockIdx.x) * 16384) + ((((int)threadIdx.y) >> 1) * 1024)) + (i2_0 * 8)) + ((((int)threadIdx.y) & 1) * 4)) + ax0_ax1_fused_3_s_1) + 8)];
      }
    }
    for (int ax0_ax1_fused_0_2 = 0; ax0_ax1_fused_0_2 < 16; ++ax0_ax1_fused_0_2) {
      B_local[((ax0_ax1_fused_0_2 * 64) + ((int)threadIdx.y))] = *(float2*)(B + ((((((i2_0 * 8192) + ((ax0_ax1_fused_0_2 >> 1) * 1024)) + (((int)blockIdx.y) * 256)) + ((ax0_ax1_fused_0_2 & 1) * 128)) + (((int)threadIdx.y) * 2)) + 8192));
    }
    __syncthreads();
    for (int i0_3 = 0; i0_3 < 16; ++i0_3) {
      for (int ax0_ax1_fused_1_s = 0; ax0_ax1_fused_1_s < 3; ++ax0_ax1_fused_1_s) {
        if (ax0_ax1_fused_1_s < 1) {
          A_local_shared_local[0] = A_local_shared[(((((i2_0 & 1) * 576) + (((i0_3 & 7) >> 2) * 64)) + ((i0_3 >> 3) * 4)) + (i0_3 & 3))];
        }
      }
      for (int ax0_ax1_fused_1_s_1 = 0; ax0_ax1_fused_1_s_1 < 4; ++ax0_ax1_fused_1_s_1) {
        if (ax0_ax1_fused_1_s_1 < 1) {
          B_local_shared_local[0] = B_local_shared[(((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3))];
          B_local_shared_local[1] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 32)];
          B_local_shared_local[2] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 128)];
          B_local_shared_local[3] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 160)];
        }
      }
      Y_local[i0_3] = (Y_local[i0_3] + (A_local_shared_local[0] * B_local_shared_local[0]));
      Y_local[(i0_3 + 16)] = (Y_local[(i0_3 + 16)] + (A_local_shared_local[0] * B_local_shared_local[1]));
      Y_local[(i0_3 + 32)] = (Y_local[(i0_3 + 32)] + (A_local_shared_local[0] * B_local_shared_local[2]));
      Y_local[(i0_3 + 48)] = (Y_local[(i0_3 + 48)] + (A_local_shared_local[0] * B_local_shared_local[3]));
      for (int ax0_ax1_fused_1_s_2 = 0; ax0_ax1_fused_1_s_2 < 3; ++ax0_ax1_fused_1_s_2) {
        if (ax0_ax1_fused_1_s_2 < 1) {
          A_local_shared_local[0] = A_local_shared[((((((i2_0 & 1) * 576) + (((i0_3 & 7) >> 2) * 64)) + ((i0_3 >> 3) * 4)) + (i0_3 & 3)) + 72)];
        }
      }
      for (int ax0_ax1_fused_1_s_3 = 0; ax0_ax1_fused_1_s_3 < 4; ++ax0_ax1_fused_1_s_3) {
        if (ax0_ax1_fused_1_s_3 < 1) {
          B_local_shared_local[0] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 256)];
          B_local_shared_local[1] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 288)];
          B_local_shared_local[2] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 384)];
          B_local_shared_local[3] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 416)];
        }
      }
      Y_local[i0_3] = (Y_local[i0_3] + (A_local_shared_local[0] * B_local_shared_local[0]));
      Y_local[(i0_3 + 16)] = (Y_local[(i0_3 + 16)] + (A_local_shared_local[0] * B_local_shared_local[1]));
      Y_local[(i0_3 + 32)] = (Y_local[(i0_3 + 32)] + (A_local_shared_local[0] * B_local_shared_local[2]));
      Y_local[(i0_3 + 48)] = (Y_local[(i0_3 + 48)] + (A_local_shared_local[0] * B_local_shared_local[3]));
    }
    for (int i0_3_1 = 0; i0_3_1 < 16; ++i0_3_1) {
      for (int ax0_ax1_fused_1_s_4 = 0; ax0_ax1_fused_1_s_4 < 3; ++ax0_ax1_fused_1_s_4) {
        if (ax0_ax1_fused_1_s_4 < 1) {
          A_local_shared_local[0] = A_local_shared[((((((i2_0 & 1) * 576) + (((i0_3_1 & 7) >> 2) * 64)) + ((i0_3_1 >> 3) * 4)) + (i0_3_1 & 3)) + 144)];
        }
      }
      for (int ax0_ax1_fused_1_s_5 = 0; ax0_ax1_fused_1_s_5 < 4; ++ax0_ax1_fused_1_s_5) {
        if (ax0_ax1_fused_1_s_5 < 1) {
          B_local_shared_local[0] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 512)];
          B_local_shared_local[1] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 544)];
          B_local_shared_local[2] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 640)];
          B_local_shared_local[3] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 672)];
        }
      }
      Y_local[i0_3_1] = (Y_local[i0_3_1] + (A_local_shared_local[0] * B_local_shared_local[0]));
      Y_local[(i0_3_1 + 16)] = (Y_local[(i0_3_1 + 16)] + (A_local_shared_local[0] * B_local_shared_local[1]));
      Y_local[(i0_3_1 + 32)] = (Y_local[(i0_3_1 + 32)] + (A_local_shared_local[0] * B_local_shared_local[2]));
      Y_local[(i0_3_1 + 48)] = (Y_local[(i0_3_1 + 48)] + (A_local_shared_local[0] * B_local_shared_local[3]));
      for (int ax0_ax1_fused_1_s_6 = 0; ax0_ax1_fused_1_s_6 < 3; ++ax0_ax1_fused_1_s_6) {
        if (ax0_ax1_fused_1_s_6 < 1) {
          A_local_shared_local[0] = A_local_shared[((((((i2_0 & 1) * 576) + (((i0_3_1 & 7) >> 2) * 64)) + ((i0_3_1 >> 3) * 4)) + (i0_3_1 & 3)) + 216)];
        }
      }
      for (int ax0_ax1_fused_1_s_7 = 0; ax0_ax1_fused_1_s_7 < 4; ++ax0_ax1_fused_1_s_7) {
        if (ax0_ax1_fused_1_s_7 < 1) {
          B_local_shared_local[0] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 768)];
          B_local_shared_local[1] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 800)];
          B_local_shared_local[2] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 896)];
          B_local_shared_local[3] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 928)];
        }
      }
      Y_local[i0_3_1] = (Y_local[i0_3_1] + (A_local_shared_local[0] * B_local_shared_local[0]));
      Y_local[(i0_3_1 + 16)] = (Y_local[(i0_3_1 + 16)] + (A_local_shared_local[0] * B_local_shared_local[1]));
      Y_local[(i0_3_1 + 32)] = (Y_local[(i0_3_1 + 32)] + (A_local_shared_local[0] * B_local_shared_local[2]));
      Y_local[(i0_3_1 + 48)] = (Y_local[(i0_3_1 + 48)] + (A_local_shared_local[0] * B_local_shared_local[3]));
    }
    for (int i0_3_2 = 0; i0_3_2 < 16; ++i0_3_2) {
      for (int ax0_ax1_fused_1_s_8 = 0; ax0_ax1_fused_1_s_8 < 3; ++ax0_ax1_fused_1_s_8) {
        if (ax0_ax1_fused_1_s_8 < 1) {
          A_local_shared_local[0] = A_local_shared[((((((i2_0 & 1) * 576) + (((i0_3_2 & 7) >> 2) * 64)) + ((i0_3_2 >> 3) * 4)) + (i0_3_2 & 3)) + 288)];
        }
      }
      for (int ax0_ax1_fused_1_s_9 = 0; ax0_ax1_fused_1_s_9 < 4; ++ax0_ax1_fused_1_s_9) {
        if (ax0_ax1_fused_1_s_9 < 1) {
          B_local_shared_local[0] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1024)];
          B_local_shared_local[1] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1056)];
          B_local_shared_local[2] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1152)];
          B_local_shared_local[3] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1184)];
        }
      }
      Y_local[i0_3_2] = (Y_local[i0_3_2] + (A_local_shared_local[0] * B_local_shared_local[0]));
      Y_local[(i0_3_2 + 16)] = (Y_local[(i0_3_2 + 16)] + (A_local_shared_local[0] * B_local_shared_local[1]));
      Y_local[(i0_3_2 + 32)] = (Y_local[(i0_3_2 + 32)] + (A_local_shared_local[0] * B_local_shared_local[2]));
      Y_local[(i0_3_2 + 48)] = (Y_local[(i0_3_2 + 48)] + (A_local_shared_local[0] * B_local_shared_local[3]));
      for (int ax0_ax1_fused_1_s_10 = 0; ax0_ax1_fused_1_s_10 < 3; ++ax0_ax1_fused_1_s_10) {
        if (ax0_ax1_fused_1_s_10 < 1) {
          A_local_shared_local[0] = A_local_shared[((((((i2_0 & 1) * 576) + (((i0_3_2 & 7) >> 2) * 64)) + ((i0_3_2 >> 3) * 4)) + (i0_3_2 & 3)) + 360)];
        }
      }
      for (int ax0_ax1_fused_1_s_11 = 0; ax0_ax1_fused_1_s_11 < 4; ++ax0_ax1_fused_1_s_11) {
        if (ax0_ax1_fused_1_s_11 < 1) {
          B_local_shared_local[0] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1280)];
          B_local_shared_local[1] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1312)];
          B_local_shared_local[2] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1408)];
          B_local_shared_local[3] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1440)];
        }
      }
      Y_local[i0_3_2] = (Y_local[i0_3_2] + (A_local_shared_local[0] * B_local_shared_local[0]));
      Y_local[(i0_3_2 + 16)] = (Y_local[(i0_3_2 + 16)] + (A_local_shared_local[0] * B_local_shared_local[1]));
      Y_local[(i0_3_2 + 32)] = (Y_local[(i0_3_2 + 32)] + (A_local_shared_local[0] * B_local_shared_local[2]));
      Y_local[(i0_3_2 + 48)] = (Y_local[(i0_3_2 + 48)] + (A_local_shared_local[0] * B_local_shared_local[3]));
    }
    for (int i0_3_3 = 0; i0_3_3 < 16; ++i0_3_3) {
      for (int ax0_ax1_fused_1_s_12 = 0; ax0_ax1_fused_1_s_12 < 3; ++ax0_ax1_fused_1_s_12) {
        if (ax0_ax1_fused_1_s_12 < 1) {
          A_local_shared_local[0] = A_local_shared[((((((i2_0 & 1) * 576) + (((i0_3_3 & 7) >> 2) * 64)) + ((i0_3_3 >> 3) * 4)) + (i0_3_3 & 3)) + 432)];
        }
      }
      for (int ax0_ax1_fused_1_s_13 = 0; ax0_ax1_fused_1_s_13 < 4; ++ax0_ax1_fused_1_s_13) {
        if (ax0_ax1_fused_1_s_13 < 1) {
          B_local_shared_local[0] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1536)];
          B_local_shared_local[1] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1568)];
          B_local_shared_local[2] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1664)];
          B_local_shared_local[3] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1696)];
        }
      }
      Y_local[i0_3_3] = (Y_local[i0_3_3] + (A_local_shared_local[0] * B_local_shared_local[0]));
      Y_local[(i0_3_3 + 16)] = (Y_local[(i0_3_3 + 16)] + (A_local_shared_local[0] * B_local_shared_local[1]));
      Y_local[(i0_3_3 + 32)] = (Y_local[(i0_3_3 + 32)] + (A_local_shared_local[0] * B_local_shared_local[2]));
      Y_local[(i0_3_3 + 48)] = (Y_local[(i0_3_3 + 48)] + (A_local_shared_local[0] * B_local_shared_local[3]));
      for (int ax0_ax1_fused_1_s_14 = 0; ax0_ax1_fused_1_s_14 < 3; ++ax0_ax1_fused_1_s_14) {
        if (ax0_ax1_fused_1_s_14 < 1) {
          A_local_shared_local[0] = A_local_shared[((((((i2_0 & 1) * 576) + (((i0_3_3 & 7) >> 2) * 64)) + ((i0_3_3 >> 3) * 4)) + (i0_3_3 & 3)) + 504)];
        }
      }
      for (int ax0_ax1_fused_1_s_15 = 0; ax0_ax1_fused_1_s_15 < 4; ++ax0_ax1_fused_1_s_15) {
        if (ax0_ax1_fused_1_s_15 < 1) {
          B_local_shared_local[0] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1792)];
          B_local_shared_local[1] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1824)];
          B_local_shared_local[2] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1920)];
          B_local_shared_local[3] = B_local_shared[((((((i2_0 & 1) * 2048) + (((((int)threadIdx.y) & 7) >> 2) * 64)) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 1952)];
        }
      }
      Y_local[i0_3_3] = (Y_local[i0_3_3] + (A_local_shared_local[0] * B_local_shared_local[0]));
      Y_local[(i0_3_3 + 16)] = (Y_local[(i0_3_3 + 16)] + (A_local_shared_local[0] * B_local_shared_local[1]));
      Y_local[(i0_3_3 + 32)] = (Y_local[(i0_3_3 + 32)] + (A_local_shared_local[0] * B_local_shared_local[2]));
      Y_local[(i0_3_3 + 48)] = (Y_local[(i0_3_3 + 48)] + (A_local_shared_local[0] * B_local_shared_local[3]));
    }
    __syncthreads();
      int2 __3 = make_int2(((((((((i2_0 + 1) & 1) * 576) + ((((int)threadIdx.y) & 3) * 144)) + (((((int)threadIdx.y) & 31) >> 4) * 64)) + ((((int)threadIdx.y) >> 5) * 4)) + ((((int)threadIdx.y) & 15) >> 2)))+(72*0), ((((((((i2_0 + 1) & 1) * 576) + ((((int)threadIdx.y) & 3) * 144)) + (((((int)threadIdx.y) & 31) >> 4) * 64)) + ((((int)threadIdx.y) >> 5) * 4)) + ((((int)threadIdx.y) & 15) >> 2)))+(72*1));
      float2 __4 = *(float2*)(A_local + (((((int)threadIdx.y) * 2) - min(((((int)threadIdx.y) & 1) * 4), ((((int)threadIdx.y) & 3) * 2))) - (min((((int)threadIdx.y) >> 1), (((int)threadIdx.y) >> 2)) * 8)));
      A_local_shared[__3.x] = __4.x;
      A_local_shared[__3.y] = __4.y;
    for (int ax0_ax1_fused_0_3 = 0; ax0_ax1_fused_0_3 < 16; ++ax0_ax1_fused_0_3) {
      *(float2*)(B_local_shared + (((((((i2_0 + 1) & 1) * 2048) + (ax0_ax1_fused_0_3 * 128)) + (((((int)threadIdx.y) & 3) >> 1) * 64)) + ((((int)threadIdx.y) >> 2) * 4)) + ((((int)threadIdx.y) & 1) * 2))) = B_local[((ax0_ax1_fused_0_3 * 64) + ((int)threadIdx.y))];
    }
  }
  __syncthreads();
  for (int i0_3_4 = 0; i0_3_4 < 16; ++i0_3_4) {
    for (int ax0_ax1_fused_1_s_16 = 0; ax0_ax1_fused_1_s_16 < 3; ++ax0_ax1_fused_1_s_16) {
      if (ax0_ax1_fused_1_s_16 < 1) {
        A_local_shared_local_1[0] = A_local_shared[((((((i0_3_4 & 7) >> 2) * 64) + ((i0_3_4 >> 3) * 4)) + (i0_3_4 & 3)) + 576)];
      }
    }
    for (int ax0_ax1_fused_1_s_17 = 0; ax0_ax1_fused_1_s_17 < 4; ++ax0_ax1_fused_1_s_17) {
      if (ax0_ax1_fused_1_s_17 < 1) {
        B_local_shared_local_1[0] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2048)];
        B_local_shared_local_1[1] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2080)];
        B_local_shared_local_1[2] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2176)];
        B_local_shared_local_1[3] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2208)];
      }
    }
    Y_local[i0_3_4] = (Y_local[i0_3_4] + (A_local_shared_local_1[0] * B_local_shared_local_1[0]));
    Y_local[(i0_3_4 + 16)] = (Y_local[(i0_3_4 + 16)] + (A_local_shared_local_1[0] * B_local_shared_local_1[1]));
    Y_local[(i0_3_4 + 32)] = (Y_local[(i0_3_4 + 32)] + (A_local_shared_local_1[0] * B_local_shared_local_1[2]));
    Y_local[(i0_3_4 + 48)] = (Y_local[(i0_3_4 + 48)] + (A_local_shared_local_1[0] * B_local_shared_local_1[3]));
    for (int ax0_ax1_fused_1_s_18 = 0; ax0_ax1_fused_1_s_18 < 3; ++ax0_ax1_fused_1_s_18) {
      if (ax0_ax1_fused_1_s_18 < 1) {
        A_local_shared_local_1[0] = A_local_shared[((((((i0_3_4 & 7) >> 2) * 64) + ((i0_3_4 >> 3) * 4)) + (i0_3_4 & 3)) + 648)];
      }
    }
    for (int ax0_ax1_fused_1_s_19 = 0; ax0_ax1_fused_1_s_19 < 4; ++ax0_ax1_fused_1_s_19) {
      if (ax0_ax1_fused_1_s_19 < 1) {
        B_local_shared_local_1[0] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2304)];
        B_local_shared_local_1[1] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2336)];
        B_local_shared_local_1[2] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2432)];
        B_local_shared_local_1[3] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2464)];
      }
    }
    Y_local[i0_3_4] = (Y_local[i0_3_4] + (A_local_shared_local_1[0] * B_local_shared_local_1[0]));
    Y_local[(i0_3_4 + 16)] = (Y_local[(i0_3_4 + 16)] + (A_local_shared_local_1[0] * B_local_shared_local_1[1]));
    Y_local[(i0_3_4 + 32)] = (Y_local[(i0_3_4 + 32)] + (A_local_shared_local_1[0] * B_local_shared_local_1[2]));
    Y_local[(i0_3_4 + 48)] = (Y_local[(i0_3_4 + 48)] + (A_local_shared_local_1[0] * B_local_shared_local_1[3]));
  }
  for (int i0_3_5 = 0; i0_3_5 < 16; ++i0_3_5) {
    for (int ax0_ax1_fused_1_s_20 = 0; ax0_ax1_fused_1_s_20 < 3; ++ax0_ax1_fused_1_s_20) {
      if (ax0_ax1_fused_1_s_20 < 1) {
        A_local_shared_local_1[0] = A_local_shared[((((((i0_3_5 & 7) >> 2) * 64) + ((i0_3_5 >> 3) * 4)) + (i0_3_5 & 3)) + 720)];
      }
    }
    for (int ax0_ax1_fused_1_s_21 = 0; ax0_ax1_fused_1_s_21 < 4; ++ax0_ax1_fused_1_s_21) {
      if (ax0_ax1_fused_1_s_21 < 1) {
        B_local_shared_local_1[0] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2560)];
        B_local_shared_local_1[1] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2592)];
        B_local_shared_local_1[2] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2688)];
        B_local_shared_local_1[3] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2720)];
      }
    }
    Y_local[i0_3_5] = (Y_local[i0_3_5] + (A_local_shared_local_1[0] * B_local_shared_local_1[0]));
    Y_local[(i0_3_5 + 16)] = (Y_local[(i0_3_5 + 16)] + (A_local_shared_local_1[0] * B_local_shared_local_1[1]));
    Y_local[(i0_3_5 + 32)] = (Y_local[(i0_3_5 + 32)] + (A_local_shared_local_1[0] * B_local_shared_local_1[2]));
    Y_local[(i0_3_5 + 48)] = (Y_local[(i0_3_5 + 48)] + (A_local_shared_local_1[0] * B_local_shared_local_1[3]));
    for (int ax0_ax1_fused_1_s_22 = 0; ax0_ax1_fused_1_s_22 < 3; ++ax0_ax1_fused_1_s_22) {
      if (ax0_ax1_fused_1_s_22 < 1) {
        A_local_shared_local_1[0] = A_local_shared[((((((i0_3_5 & 7) >> 2) * 64) + ((i0_3_5 >> 3) * 4)) + (i0_3_5 & 3)) + 792)];
      }
    }
    for (int ax0_ax1_fused_1_s_23 = 0; ax0_ax1_fused_1_s_23 < 4; ++ax0_ax1_fused_1_s_23) {
      if (ax0_ax1_fused_1_s_23 < 1) {
        B_local_shared_local_1[0] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2816)];
        B_local_shared_local_1[1] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2848)];
        B_local_shared_local_1[2] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2944)];
        B_local_shared_local_1[3] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 2976)];
      }
    }
    Y_local[i0_3_5] = (Y_local[i0_3_5] + (A_local_shared_local_1[0] * B_local_shared_local_1[0]));
    Y_local[(i0_3_5 + 16)] = (Y_local[(i0_3_5 + 16)] + (A_local_shared_local_1[0] * B_local_shared_local_1[1]));
    Y_local[(i0_3_5 + 32)] = (Y_local[(i0_3_5 + 32)] + (A_local_shared_local_1[0] * B_local_shared_local_1[2]));
    Y_local[(i0_3_5 + 48)] = (Y_local[(i0_3_5 + 48)] + (A_local_shared_local_1[0] * B_local_shared_local_1[3]));
  }
  for (int i0_3_6 = 0; i0_3_6 < 16; ++i0_3_6) {
    for (int ax0_ax1_fused_1_s_24 = 0; ax0_ax1_fused_1_s_24 < 3; ++ax0_ax1_fused_1_s_24) {
      if (ax0_ax1_fused_1_s_24 < 1) {
        A_local_shared_local_1[0] = A_local_shared[((((((i0_3_6 & 7) >> 2) * 64) + ((i0_3_6 >> 3) * 4)) + (i0_3_6 & 3)) + 864)];
      }
    }
    for (int ax0_ax1_fused_1_s_25 = 0; ax0_ax1_fused_1_s_25 < 4; ++ax0_ax1_fused_1_s_25) {
      if (ax0_ax1_fused_1_s_25 < 1) {
        B_local_shared_local_1[0] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3072)];
        B_local_shared_local_1[1] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3104)];
        B_local_shared_local_1[2] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3200)];
        B_local_shared_local_1[3] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3232)];
      }
    }
    Y_local[i0_3_6] = (Y_local[i0_3_6] + (A_local_shared_local_1[0] * B_local_shared_local_1[0]));
    Y_local[(i0_3_6 + 16)] = (Y_local[(i0_3_6 + 16)] + (A_local_shared_local_1[0] * B_local_shared_local_1[1]));
    Y_local[(i0_3_6 + 32)] = (Y_local[(i0_3_6 + 32)] + (A_local_shared_local_1[0] * B_local_shared_local_1[2]));
    Y_local[(i0_3_6 + 48)] = (Y_local[(i0_3_6 + 48)] + (A_local_shared_local_1[0] * B_local_shared_local_1[3]));
    for (int ax0_ax1_fused_1_s_26 = 0; ax0_ax1_fused_1_s_26 < 3; ++ax0_ax1_fused_1_s_26) {
      if (ax0_ax1_fused_1_s_26 < 1) {
        A_local_shared_local_1[0] = A_local_shared[((((((i0_3_6 & 7) >> 2) * 64) + ((i0_3_6 >> 3) * 4)) + (i0_3_6 & 3)) + 936)];
      }
    }
    for (int ax0_ax1_fused_1_s_27 = 0; ax0_ax1_fused_1_s_27 < 4; ++ax0_ax1_fused_1_s_27) {
      if (ax0_ax1_fused_1_s_27 < 1) {
        B_local_shared_local_1[0] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3328)];
        B_local_shared_local_1[1] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3360)];
        B_local_shared_local_1[2] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3456)];
        B_local_shared_local_1[3] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3488)];
      }
    }
    Y_local[i0_3_6] = (Y_local[i0_3_6] + (A_local_shared_local_1[0] * B_local_shared_local_1[0]));
    Y_local[(i0_3_6 + 16)] = (Y_local[(i0_3_6 + 16)] + (A_local_shared_local_1[0] * B_local_shared_local_1[1]));
    Y_local[(i0_3_6 + 32)] = (Y_local[(i0_3_6 + 32)] + (A_local_shared_local_1[0] * B_local_shared_local_1[2]));
    Y_local[(i0_3_6 + 48)] = (Y_local[(i0_3_6 + 48)] + (A_local_shared_local_1[0] * B_local_shared_local_1[3]));
  }
  for (int i0_3_7 = 0; i0_3_7 < 16; ++i0_3_7) {
    for (int ax0_ax1_fused_1_s_28 = 0; ax0_ax1_fused_1_s_28 < 3; ++ax0_ax1_fused_1_s_28) {
      if (ax0_ax1_fused_1_s_28 < 1) {
        A_local_shared_local_1[0] = A_local_shared[((((((i0_3_7 & 7) >> 2) * 64) + ((i0_3_7 >> 3) * 4)) + (i0_3_7 & 3)) + 1008)];
      }
    }
    for (int ax0_ax1_fused_1_s_29 = 0; ax0_ax1_fused_1_s_29 < 4; ++ax0_ax1_fused_1_s_29) {
      if (ax0_ax1_fused_1_s_29 < 1) {
        B_local_shared_local_1[0] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3584)];
        B_local_shared_local_1[1] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3616)];
        B_local_shared_local_1[2] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3712)];
        B_local_shared_local_1[3] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3744)];
      }
    }
    Y_local[i0_3_7] = (Y_local[i0_3_7] + (A_local_shared_local_1[0] * B_local_shared_local_1[0]));
    Y_local[(i0_3_7 + 16)] = (Y_local[(i0_3_7 + 16)] + (A_local_shared_local_1[0] * B_local_shared_local_1[1]));
    Y_local[(i0_3_7 + 32)] = (Y_local[(i0_3_7 + 32)] + (A_local_shared_local_1[0] * B_local_shared_local_1[2]));
    Y_local[(i0_3_7 + 48)] = (Y_local[(i0_3_7 + 48)] + (A_local_shared_local_1[0] * B_local_shared_local_1[3]));
    for (int ax0_ax1_fused_1_s_30 = 0; ax0_ax1_fused_1_s_30 < 3; ++ax0_ax1_fused_1_s_30) {
      if (ax0_ax1_fused_1_s_30 < 1) {
        A_local_shared_local_1[0] = A_local_shared[((((((i0_3_7 & 7) >> 2) * 64) + ((i0_3_7 >> 3) * 4)) + (i0_3_7 & 3)) + 1080)];
      }
    }
    for (int ax0_ax1_fused_1_s_31 = 0; ax0_ax1_fused_1_s_31 < 4; ++ax0_ax1_fused_1_s_31) {
      if (ax0_ax1_fused_1_s_31 < 1) {
        B_local_shared_local_1[0] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3840)];
        B_local_shared_local_1[1] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3872)];
        B_local_shared_local_1[2] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 3968)];
        B_local_shared_local_1[3] = B_local_shared[((((((((int)threadIdx.y) & 7) >> 2) * 64) + ((((int)threadIdx.y) >> 3) * 4)) + (((int)threadIdx.y) & 3)) + 4000)];
      }
    }
    Y_local[i0_3_7] = (Y_local[i0_3_7] + (A_local_shared_local_1[0] * B_local_shared_local_1[0]));
    Y_local[(i0_3_7 + 16)] = (Y_local[(i0_3_7 + 16)] + (A_local_shared_local_1[0] * B_local_shared_local_1[1]));
    Y_local[(i0_3_7 + 32)] = (Y_local[(i0_3_7 + 32)] + (A_local_shared_local_1[0] * B_local_shared_local_1[2]));
    Y_local[(i0_3_7 + 48)] = (Y_local[(i0_3_7 + 48)] + (A_local_shared_local_1[0] * B_local_shared_local_1[3]));
  }
  for (int ax0 = 0; ax0 < 16; ++ax0) {
    Y[((((((int)blockIdx.x) * 16384) + (ax0 * 1024)) + (((int)blockIdx.y) * 256)) + ((int)threadIdx.y))] = Y_local[ax0];
    Y[(((((((int)blockIdx.x) * 16384) + (ax0 * 1024)) + (((int)blockIdx.y) * 256)) + ((int)threadIdx.y)) + 64)] = Y_local[(ax0 + 16)];
    Y[(((((((int)blockIdx.x) * 16384) + (ax0 * 1024)) + (((int)blockIdx.y) * 256)) + ((int)threadIdx.y)) + 128)] = Y_local[(ax0 + 32)];
    Y[(((((((int)blockIdx.x) * 16384) + (ax0 * 1024)) + (((int)blockIdx.y) * 256)) + ((int)threadIdx.y)) + 192)] = Y_local[(ax0 + 48)];
  }
}


